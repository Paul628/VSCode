#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>

inline void GPUassert(hipError_t code, char * file, int line, bool Abort=true)
{
    if (code != 0) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),file,line);
        if (Abort) exit(code);
    }       
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__host__ __device__ void swap(char *x, char *y)
{
    char temp;
    temp = *x;
    *x = *y;
    *y = temp;
}

__device__ void permute_device(char *a, int l, int r, int tid, int* count, int n)
{
    if (l == r) {
        // Print the permutation
        //printf("Count: %d  Thread %d: Permutation: %s\n",*count, tid, a);
        //for (int i = 0; i < n; i++) printf("%c", a[i]);
        //printf("\n");
        (*count)++;
    } else {
        for (int i = l; i <= r; i++) {
            swap(&a[l], &a[i]);
            permute_device(a, l + 1, r, tid, count, n);
            swap(&a[l], &a[i]); // backtrack
        }
    }
}

__global__ void permute_kernel(char* d_A, int n, int* d_counts)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;

    char* local_array = new char[n];
    for (int i = 0; i < n; i++) local_array[i] = d_A[i];
    swap(&local_array[0], &local_array[tid]);

    int count = 0;
    permute_device(local_array, 1, n - 1, tid, &count, n);

    d_counts[tid] = count; // Store each thread's count
    delete[] local_array;
}

int main()
{
    char h_a[] = "ABCDEFGHIJK"; // Give string here. TODO: Make set in the end because recurring characters reduce unique permutations
    int n = strlen(h_a); //Dynamic for string length

    printf("\nSize of string: %d\n", n);

    char* d_a;
    GPUerrchk(hipMalloc((void**)&d_a, n * sizeof(char)));
    GPUerrchk(hipMemcpy(d_a, h_a, n * sizeof(char), hipMemcpyHostToDevice));

    // Allocate device and host memory for counts
    int* d_counts;
    int* h_counts = (int*)malloc(n * sizeof(int));
    GPUerrchk(hipMalloc((void**)&d_counts, n * sizeof(int)));

    printf("\nGenerating permutations on GPU\n");
    double start = clock();
    permute_kernel<<<1, n>>>(d_a, n, d_counts);
    GPUerrchk(hipPeekAtLastError());
    GPUerrchk(hipDeviceSynchronize());
    double end = clock();

    // Copy counts back and sum
    GPUerrchk(hipMemcpy(h_counts, d_counts, n * sizeof(int), hipMemcpyDeviceToHost));
    int total = 0;
    for (int i = 0; i < n; i++) total += h_counts[i];

    hipFree(d_a);
    hipFree(d_counts);
    free(h_counts);

    printf("Total number of permutations: %'d\n", total);
    printf("Time taken: %f seconds\n", (end - start) / CLOCKS_PER_SEC);
    getchar();
    return 0;
}