#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <windows.h>

inline void GPUassert(hipError_t code, const char *file, int line, bool Abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (Abort) exit(code);
    }
}

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

__host__ __device__ void swap(char *x, char *y) {
    char temp = *x;
    *x = *y;
    *y = temp;
}

__device__ void permute_device(char *a, int i, int n, int tid, int* count) {
    if (i == n) {
        char* c = a - 1; // Points to the start of the original array
        //printf("Permutation nr. %i from thread nr. %i: %s\n", *count, tid, c);
        count[0]++;
    } 
    else {
        for (int j = i; j <= n; j++) {
            swap(a + i, a + j);
            permute_device(a, i + 1, n, tid, count);
            swap(a + i, a + j); // backtrack
        }
    }
}

__global__ void permute_kernel(char* d_A, int size, int* d_counts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int permute_length = size - 1; // Exclude null terminator
    if (tid >= permute_length) return;

    int count = 0;
    char* local_array = new char[size];

    // Copy from global to local memory
    for (int i = 0; i < size; i++) {
        local_array[i] = d_A[i];
    }

    swap(&local_array[0], &local_array[tid]);
    permute_device(local_array + 1, 0, permute_length - 2, tid, &count);
    d_counts[tid] = count; // Store each thread's count
    delete[] local_array;
}

void print_with_separator(int num, char sep) {
    char buffer[50];
    sprintf(buffer, "%d", num); // Convert to string
    int len = strlen(buffer);
    int pos = len % 3 == 0 ? 3 : len % 3; // Position for first separator

    for (int i = 0; i < len; i++) {
        putchar(buffer[i]);
        if ((i + 1 - pos) % 3 == 0 && i != len - 1) {
            putchar(sep); // Insert separator
        }
    }
}

int main() {

    char h_a[] = "ABCDEFGHIJK"; // Example input
    int permute_length = strlen(h_a);
    size_t total_size = permute_length + 1; // Include null terminator

    printf("String length: %d, \n", permute_length);

    char* d_a;
    GPUerrchk(hipMalloc((void**)&d_a, total_size));
    GPUerrchk(hipMemcpy(d_a, h_a, total_size, hipMemcpyHostToDevice));

    // Allocate device and host memory for counts
    int* d_counts;
    int* h_counts = (int*)malloc(permute_length * sizeof(int));
    GPUerrchk(hipMalloc((void**)&d_counts, permute_length * sizeof(int)));

    printf("\nPermutations on GPU:\n");
    double start = clock();
    permute_kernel<<<1, permute_length>>>(d_a, total_size, d_counts);
    GPUerrchk(hipPeekAtLastError());
    GPUerrchk(hipDeviceSynchronize());
    double end = clock();

    // Get counts and sum them up
    GPUerrchk(hipMemcpy(h_counts, d_counts, permute_length * sizeof(int), hipMemcpyDeviceToHost));
    int total = 0;
    for (int i = 0; i < permute_length; i++) total += h_counts[i];
    
    printf("Total number of permutations: ");
    print_with_separator(total, '.');
    printf("\nTime taken: %f seconds\n", (end - start) / CLOCKS_PER_SEC);

    hipFree(d_a);
    hipFree(d_counts);
    free(h_counts);
    getchar();
    return 0;
}