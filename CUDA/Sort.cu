#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <time.h>

#define N 10000000  // Number of elements to sort
#define CHUNK 1024  // Each block sorts CHUNK elements

__device__ void insertion_sort(int* data, int left, int right) {
    for (int i = left + 1; i <= right; i++) {
        int key = data[i];
        int j = i - 1;
        while (j >= left && data[j] > key) {
            data[j + 1] = data[j];
            j--;
        }
        data[j + 1] = key;
    }
}

__global__ void sort_chunks(int* d_arr, int n) {
    //int idx = blockIdx.x * blockDim.x;
    int idx = blockIdx.x * CHUNK;
    int left = idx;
    int right = min(idx + CHUNK - 1, n - 1);
    if (left < n) {
        insertion_sort(d_arr, left, right);
    }
}

// Host merge for two sorted subarrays
void merge(int* arr, int left, int mid, int right, int* temp) {
    int i = left, j = mid + 1, k = left;
    while (i <= mid && j <= right) {
        if (arr[i] <= arr[j]) temp[k++] = arr[i++];
        else temp[k++] = arr[j++];
    }
    while (i <= mid) temp[k++] = arr[i++];
    while (j <= right) temp[k++] = arr[j++];
    for (int l = left; l <= right; l++) arr[l] = temp[l];
}

void swap (int *a, int *b)
{
    int temp = *a;
    *a = *b;
    *b = temp;
}

void randomize ( int arr[], int n )
{
    // Use a different seed value so that we don't get same
    // result each time we run this program
    srand ( time(NULL) );

    // Start from the last element and swap one by one. We don't
    // need to run for the first element that's why i > 0
    for (int i = n-1; i > 0; i--)
    {
        // Pick a random index from 0 to i
        int j = rand() % (i+1);

        // Swap arr[i] with the element at random index
        swap(&arr[i], &arr[j]);
    }
}

void print_with_separator(int num, char sep) {
    char buffer[50];
    sprintf(buffer, "%d", num); // Convert to string
    int len = strlen(buffer);
    int pos = len % 3 == 0 ? 3 : len % 3; // Position for first separator

    for (int i = 0; i < len; i++) {
        putchar(buffer[i]);
        if ((i + 1 - pos) % 3 == 0 && i != len - 1) {
            putchar(sep); // Insert separator
        }
    }
}

int main() {
    int* h_arr = (int*)malloc(N * sizeof(int));
    int* temp = (int*)malloc(N * sizeof(int));
    int n = sizeof(h_arr)/ sizeof(h_arr[0]);
    for (int i = 0; i < N; i++) h_arr[i] = i;
    randomize(h_arr, n);

    int* d_arr;
    hipMalloc(&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    int num_blocks = (N + CHUNK - 1) / CHUNK;

    printf("Sorting: ");
    print_with_separator(N, '.');
    printf(" Elements\n");
    clock_t start = clock();
    sort_chunks<<<num_blocks, 1>>>(d_arr, N);
    hipDeviceSynchronize();
    clock_t end = clock();
    printf("GPU chunk sort time: %.2f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    // Iterative merge on host
    start = clock();
    for (int size = CHUNK; size < N; size *= 2) {
        for (int left = 0; left < N; left += 2 * size) {
            int mid = std::min(left + size - 1, N - 1);
            int right = std::min(left + 2 * size - 1, N - 1);
            if (mid < right)
                merge(h_arr, left, mid, right, temp);
        }
    }
    end = clock();
    printf("Host merge time: %.2f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

    // Optional: check if sorted
    bool sorted = true;
    for (int i = 1; i < N; i++) {
        if (h_arr[i-1] > h_arr[i]) {
            sorted = false;
            break;
        }
    }
    printf("Sorted: %s\n", sorted ? "True" : "False");

    hipFree(d_arr);
    free(h_arr);
    free(temp);
    getchar();
    return 0;
}